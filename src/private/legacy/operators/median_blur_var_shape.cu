#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION. All rights reserved.
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cv_cuda.h"
#include "cuda_utils.cuh"
#include "border.cuh"
#include "cv_utils.h"
#include "hip/hip_vector_types.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"


#define GENERAL_KERNEL_BLOCK 32
#define SMALL_KERNEL_BLOCK 16
using namespace cv::cudev;
using namespace cuda_op;

/**
 * This function fetches the pixel from the shared if possible.
 * Otherwise, the pixel is read from global memory.
 * If the given index is out of bound, then based on the rule of cv::BORDER_REPLICATE,
 * this function fetches the nearest valid pixel.
 * @tparam T The type of the pixels stored.
 * @param shared a pointer of type T to shared memory,
 * @param src a Ptr2dNHWC <T> stored in global memory.
 * @param batchIdx the index of the image.
 * @param h the height of the image.
 * @param w the width of the image.
 * @param c the channel being processed.
 * @param sxOffset the x offset that is subtracted from gx to calculate the corresponding pixel index in shared memory.
 * @param syOffset the y offset that is subtracted from gy to calculate the corresponding pixel index in shared memory.
 * @param gx the horizontal index of the desired pixel in the image.
 * @param gy the vertical index of the desired pixel in the image.
 * @return the pixel at given index
 */
template<typename T>
__device__ T fetch(T *shared, const Ptr2dVarShapeNHWC <T> src, int batchIdx, int h, int w, int c,
                   int sxOffset, int syOffset, int gx, int gy, int block_size)
{
    // check for cv::BORDER_REPLICATE.
    if(gx < 0)
    {
        gx = 0;
    }
    if(gx >= w)
    {
        gx = w - 1;
    }
    if(gy < 0)
    {
        gy = 0;
    }
    if(gy >= h)
    {
        gy = h - 1;
    }
    // check if the desired pixel is not in shared memory.
    if(gy - syOffset < 0 || gy - syOffset >= blockDim.y ||
            gx - sxOffset < 0 || gx - sxOffset >= blockDim.x)
    {
        return *src.ptr(batchIdx, gy, gx, c); // fetch from global memory.
    }
    else
    {
        return shared[(gy - syOffset) * block_size + gx - sxOffset];  // fetch from shared memory.
    }
}


/**
 * Perform median fliter on the image
 * @tparam T The type of the pixels stored.
 * @param src a Ptr2dNHWC <T> stored in global memory.
 * @param dst a Ptr2dNHWC <T> stored in global memory.
 * @param kWidth width of the kernel.
 * @param kHeight height of the kernel.
 */
template<typename T>
__global__ void median(const Ptr2dVarShapeNHWC <T> src, Ptr2dVarShapeNHWC <T> dst,
                       const int *kWidths, const int *kHeights)
{
#define fetch_(gx, gy, block_size) fetch<T>(tails, src, batchIdx, h, w, channel, blockX, blockY, (gx), (gy), (block_size))
#define fetchAs1d(idx, block_size) fetch_(x - (kWidth / 2) + ((idx) % kWidth),\
        y - (kHeight / 2) + ((idx) / kWidth), (block_size))
    int tx = threadIdx.x, ty = threadIdx.y;
    int blockX = blockIdx.x * blockDim.x;
    int blockY = blockIdx.y * blockDim.y;
    int x = blockX + threadIdx.x;
    int y = blockY + threadIdx.y;
    int channel = blockIdx.z % dst.ch;
    int batchIdx = blockIdx.z / dst.ch;
    int h = src.rows[batchIdx], w = src.cols[batchIdx];
    int kWidth = kWidths[batchIdx];
    int kHeight = kHeights[batchIdx];

    __shared__ T tails[GENERAL_KERNEL_BLOCK * GENERAL_KERNEL_BLOCK];
    if(x < w && y < h)
    {
        tails[ty * GENERAL_KERNEL_BLOCK + tx] = *src.ptr(batchIdx, y, x, channel);
    }
    __syncthreads();

    if((x < w && y < h))
    {
        // min_ and max_ set up a range that we are looking for
        // only elements in that range could be median
        T tmp, pivot0, pivot1, pivot2, min_, max_;
        // In the 1st and possibly several following iterations, min_ or max_ is not assigned.
        // use isMinReady and isMaxReady to control from comparison on them.
        bool isMinReady = false, isMaxReady = false;
        int numOfEq = 0, numOfGt = 0, numOfLt = 0, numOfTaken = 0;
        int median = (kWidth * kHeight) / 2;
        int start = 0, end = kWidth * kHeight, t;
        bool isAllPreviousOutOfRange = true;

        // loop until we rule out all possible elements, and the last pivot is the median.
        while(numOfTaken < (kWidth * kHeight))
        {
            pivot0 = fetchAs1d(start, GENERAL_KERNEL_BLOCK);
            while((isMinReady && (min_ >= pivot0)) ||
                    (isMaxReady && (max_ <= pivot0)))
            {
                start++;
                pivot0 = fetchAs1d(start, GENERAL_KERNEL_BLOCK);
            }

            pivot2 = fetchAs1d(end - 1, GENERAL_KERNEL_BLOCK);
            while((isMinReady && (min_ >= pivot2)) ||
                    (isMaxReady && (max_ <= pivot2)))
            {
                end--;
                pivot2 = fetchAs1d(end - 1, GENERAL_KERNEL_BLOCK);
            }

            int idx = (start + end) / 2;
            pivot1 = fetchAs1d(idx, GENERAL_KERNEL_BLOCK);
            // check if the pivot is in the range defined by min_ and max_.
            // if not, go to the next until we find one that is in the range.
            while((isMinReady && (min_ >= pivot1)) ||
                    (isMaxReady && (max_ <= pivot1)))
            {
                idx++;
                if(idx >= end)
                {
                    idx = start;
                }
                pivot1 = fetchAs1d(idx, GENERAL_KERNEL_BLOCK);
            }

            if(pivot0 < pivot1 && pivot1 < pivot2)
            {
                pivot0 = pivot1;
            }
            else if(pivot0 < pivot2 && pivot2 < pivot1)
            {
                pivot0 = pivot2;
            }

            // use the pivot to partition the array.
            t = end;
            for(int i = start; i < t; i++)
            {
                tmp = fetchAs1d(i, GENERAL_KERNEL_BLOCK);
                // only consider the element in the range defined by min_ and max_.
                // because others are already ruled out.
                if((!isMinReady || min_ < tmp) && (!isMaxReady || tmp < max_))
                {
                    if(tmp > pivot0)
                    {
                        numOfGt++;
                    }
                    else if(tmp < pivot0)
                    {
                        numOfLt++;
                    }
                    else
                    {
                        numOfEq++;
                    }
                    if(isAllPreviousOutOfRange)
                    {
                        start = i;
                        isAllPreviousOutOfRange = false;
                    }
                    end = i + 1;
                }
            }

            // if the index of median is less than numOfLt,
            // use max_ to rule out elements greater than or equal to pivot.
            if(median < numOfLt)
            {
                max_ = pivot0;
                numOfTaken = numOfTaken + numOfEq + numOfGt;
                isMaxReady = true;
                // if the index of median is in between numOfLt and (numOfLt + numOfEq).
                // the median is found. we are lucky:)
            }
            else if(median < (numOfLt + numOfEq))
            {
                break;
                // if the index of median is greater than (numOfLt + numOfEq),
                // use min_ to rule out elements greater than or equal to pivot.
            }
            else
            {
                min_ = pivot0;
                median = median - numOfLt - numOfEq;
                numOfTaken = numOfTaken + numOfLt + numOfEq;
                isMinReady = true;
            }
            numOfLt = 0;
            numOfEq = 0;
            numOfGt = 0;
        }
        *dst.ptr(batchIdx, y, x, channel) = pivot0;
    }
}

template<typename T>
__device__ int partition(T *arr, int length, T pvt, int *numOfEq)
{
    T val;
    *numOfEq = 1;
    int i = 1;
    for(int j = 1; j < length; j++)
    {
        val = arr[j];
        if(val == pvt)
        {
            (*numOfEq) += 1;
        }
        if(val < pvt)
        {
            arr[j] = arr[i];
            arr[i] = val;
            i += 1;
        }
    }
    val = arr[0];
    arr[0] = arr[i - 1];
    arr[i - 1] = val;
    return i - 1;
}

template<typename T>
__inline__ __device__ T placePivot(T *arr, int length)
{
    int mid = length / 2;
    T pivot0 = arr[0], pivot1 = arr[mid], pivot2 = arr[length - 1];
    if(pivot0 < pivot1 && pivot1 <= pivot2)
    {
        arr[0] = pivot1;
        arr[mid] = pivot0;
        return pivot1;
    }
    if(pivot0 < pivot2 && pivot2 <= pivot1)
    {
        arr[0] = pivot2;
        arr[length - 1] = pivot0;
        return pivot2;
    }
    return pivot0;
}

template<typename T>
__global__ void medianForSmallKernel(const Ptr2dVarShapeNHWC <T> src, Ptr2dVarShapeNHWC <T> dst,
                                     const int *kWidths, const int *kHeights)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int blockX = blockIdx.x * blockDim.x;
    int blockY = blockIdx.y * blockDim.y;
    int x = blockX + threadIdx.x;
    int y = blockY + threadIdx.y;
    int channel = blockIdx.z % dst.ch;
    int batchIdx = blockIdx.z / dst.ch;
    int h = src.rows[batchIdx], w = src.cols[batchIdx];
    int kWidth = kWidths[batchIdx];
    int kHeight = kHeights[batchIdx];

    __shared__ T tails[SMALL_KERNEL_BLOCK * SMALL_KERNEL_BLOCK];
    if(x < w && y < h)
    {
        tails[ty * SMALL_KERNEL_BLOCK + tx] = *src.ptr(batchIdx, y, x, channel);
    }
    __syncthreads();

    extern  __shared__ char _arrays[];
    int length = kWidth * kHeight;
    T *arr = ((T *) _arrays) + ((tx * SMALL_KERNEL_BLOCK) + ty) * length;
    T pivot;
    int numOfEq, k = length / 2;

    if((x < w && y < h))
    {
        for(int i = 0; i < length; i++)
        {
            arr[i] = fetchAs1d(i, SMALL_KERNEL_BLOCK);
        }
        while(length > 1)
        {
            pivot = placePivot(arr, length);
            int middle = partition(arr, length, pivot, &numOfEq);
            if(k < middle)
            {
                length = middle;
            }
            else if(k < (middle + numOfEq))
            {
                *dst.ptr(batchIdx, y, x, channel) = pivot;
                return;
            }
            else
            {
                k = k - middle - 1;
                length = length - middle - 1;
                arr = arr + middle + 1;
            }
        }
        *dst.ptr(batchIdx, y, x, channel) = arr[0];
    }
}


#undef fetch_
#undef fetchAs1d

template<typename T>
void median(const void **input, void **output, const int *height,
            const int *width, int max_height, int max_width, int batch, int channels,
            int *kWidths, int *kHeights, int maxKWidth, int maxKHeight, hipStream_t stream)
{
    Ptr2dVarShapeNHWC <T> src(batch, height, width, channels, (T **) input);
    Ptr2dVarShapeNHWC <T> dst(batch, height, width, channels, (T **) output);

    #ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
    #endif

    long unsigned int sharedMemSize = SMALL_KERNEL_BLOCK * SMALL_KERNEL_BLOCK * maxKWidth * maxKHeight * sizeof(T);
    if(sharedMemSize < 48 * 1024)
    {
        dim3 block(SMALL_KERNEL_BLOCK, SMALL_KERNEL_BLOCK);
        dim3 grid(divUp(max_width, block.x), divUp(max_height, block.y), dst.ch * dst.batches);
        medianForSmallKernel<T><<<grid, block, sharedMemSize, stream>>>(src, dst, kWidths, kHeights);
        checkKernelErrors();
    }
    else
    {
        dim3 block(GENERAL_KERNEL_BLOCK, GENERAL_KERNEL_BLOCK);
        dim3 grid(divUp(max_width, block.x), divUp(max_height, block.y), dst.ch * dst.batches);
        median<T><<<grid, block, 0, stream>>>(src, dst, kWidths, kHeights);
        checkKernelErrors();
    }

    #ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
    #endif
}

namespace cuda_op
{

size_t MedianBlurVarShape::calBufferSize(int batch_size)
{
    return (2 * sizeof(void *) + 4 * sizeof(int)) * batch_size;
}

int MedianBlurVarShape::infer(const void **data_in, void **data_out, void *gpu_workspace, void *cpu_workspace,
                              const int batch, const size_t buffer_size, cv::Size *ksize, DataShape *inputShape,
                              DataFormat format, DataType data_type, hipStream_t stream)
{
    if(!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if(!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    const int channels = inputShape[0].C;

    if(channels > 4)
    {
        printf("Invalid channel number %d\n", channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    const void **inputs = (const void **) cpu_workspace;
    void **outputs = (void **)((char *) inputs + sizeof(void *) * batch);
    int *rows = (int *)((char *) outputs + sizeof(void *) * batch);
    int *cols = (int *)((char *) rows + sizeof(int) * batch);
    int *k_width = (int *)((char *) cols + sizeof(int) * batch);
    int *k_height = (int *)((char *) k_width + sizeof(int) * batch);

    size_t data_size = DataSize(data_type);
    int max_height = 0, max_width = 0, max_kHeight = 0, max_kWidth = 0;

    for(int b = 0; b < batch; b++)
    {
        if(channels != inputShape[b].C)
        {
            LOG_ERROR("Invalid Input");
            return ErrorCode::INVALID_DATA_SHAPE;
        }
        inputs[b] = data_in[b];
        outputs[b] = data_out[b];
        rows[b] = inputShape[b].H;
        cols[b] = inputShape[b].W;

        if(!(ksize[b].width > 0 && ksize[b].width % 2 == 1 && ksize[b].height > 0 && ksize[b].height % 2 == 1))
        {
            LOG_ERROR("Invalid ksize " << ksize[b].width << " " << ksize[b].height);
            return ErrorCode::INVALID_PARAMETER;
        }

        k_width[b] = ksize[b].width;
        k_height[b] = ksize[b].height;

        if(cols[b] > max_width)
            max_width = cols[b];
        if(rows[b] > max_height)
            max_height = rows[b];
        if(k_width[b]>max_kWidth)
            max_kWidth = k_width[b];
        if(k_height[b]>max_kHeight)
            max_kHeight = k_height[b];
    }

    const void **inputs_gpu = (const void **) gpu_workspace;
    void **outputs_gpu = (void **)((char *) inputs_gpu + sizeof(void *) * batch);
    int *rows_gpu = (int *)((char *) outputs_gpu + sizeof(void *) * batch);
    int *cols_gpu = (int *)((char *) rows_gpu + sizeof(int) * batch);
    int *k_width_gpu = (int *)((char *) cols_gpu + sizeof(int) * batch);
    int *k_height_gpu = (int *)((char *) k_width_gpu + sizeof(int) * batch);

    checkCudaErrors(
                    hipMemcpyAsync((void *) gpu_workspace, (void *) cpu_workspace, buffer_size, hipMemcpyHostToDevice,
                                    stream));

    typedef void (*median_t)(const void **input, void **output, const int *height,
                             const int *width, int max_height, int max_width, int batch,
                             int channels, int *kWidths, int *kHeights, int maxKWidth, int maxKHeight,
                             hipStream_t stream);

    static const median_t funcs[6] =
    {
        median<uchar>, 0, median<ushort>, 0, median<int>, median<float>,

    };
    funcs[data_type](inputs_gpu, outputs_gpu, rows_gpu, cols_gpu, max_height, max_width,
                     batch, channels, k_width_gpu, k_height_gpu, max_kWidth, max_kHeight, stream);
    return 0;
}

} // cuda_op
#undef GENERAL_KERNEL_BLOCK
#undef SMALL_KERNEL_BLOCK

#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
 * Copyright (C) 2009-2010, Willow Garage Inc., all rights reserved.
 * Copyright (C) 2014-2015, Itseez Inc., all rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

namespace nvcv::legacy::cuda_op {

static __device__ __forceinline__ float norm1(const float &a)
{
    return std::abs(a);
}

static __device__ __forceinline__ float norm1(const float2 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y);
}

static __device__ __forceinline__ float norm1(const float3 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y) + cuda::abs(a.z);
}

static __device__ __forceinline__ float norm1(const float4 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y) + cuda::abs(a.z) + cuda::abs(a.w);
}

template<typename T, typename BrdRd>
__global__ void JointBilateralFilterVarShapeKernel(const BrdRd src, const BrdRd srcColor, Ptr2dVarShapeNHWC<T> dst,
                                                   const cuda::Tensor1DWrap<int>   inDiameter,
                                                   const cuda::Tensor1DWrap<float> inSigmaColor,
                                                   const cuda::Tensor1DWrap<float> inSigmaSpace)
{
    const int batch_idx = get_batch_idx();
    const int rows      = dst.at_rows(batch_idx);
    const int columns   = dst.at_cols(batch_idx);

    // Preprocessing moved here because tensors are GPU resident
    float sigmaColor = *inSigmaColor.ptr(batch_idx);
    if (sigmaColor <= 0)
    {
        sigmaColor = 1;
    }
    float sigmaSpace = *inSigmaSpace.ptr(batch_idx);
    if (sigmaSpace <= 0)
    {
        sigmaSpace = 1;
    }

    int radius;
    int diameter = *inDiameter.ptr(batch_idx);
    if (diameter <= 0)
    {
        radius = std::roundf(sigmaSpace * 1.5f);
    }
    else
    {
        radius = diameter / 2;
    }
    if (radius < 1)
    {
        radius = 1;
    }
    assert(radius < 10000);

    const int colIdx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    const int rowIdx = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    using work_type  = cuda::ConvertBaseTypeTo<float, T>;
    int3      coord0{colIdx, rowIdx, batch_idx};
    int3      coord1{colIdx + 1, rowIdx, batch_idx};
    int3      coord2{colIdx, rowIdx + 1, batch_idx};
    int3      coord3{colIdx + 1, rowIdx + 1, batch_idx};
    work_type centerColor0 = cuda::StaticCast<float>(srcColor(coord0.z, coord0.y, coord0.x));
    work_type centerColor1 = cuda::StaticCast<float>(srcColor(coord1.z, coord1.y, coord1.x));
    work_type centerColor2 = cuda::StaticCast<float>(srcColor(coord2.z, coord2.y, coord2.x));
    work_type centerColor3 = cuda::StaticCast<float>(srcColor(coord3.z, coord3.y, coord3.x));

    int       squared_radius    = radius * radius;
    float     space_coefficient = -1 / (2 * sigmaSpace * sigmaSpace);
    float     color_coefficient = -1 / (2 * sigmaColor * sigmaColor);
    work_type numerator0        = cuda::SetAll<work_type>(0);
    work_type numerator1        = cuda::SetAll<work_type>(0);
    work_type numerator2        = cuda::SetAll<work_type>(0);
    work_type numerator3        = cuda::SetAll<work_type>(0);
    float     denominator0      = 0;
    float     denominator1      = 0;
    float     denominator2      = 0;
    float     denominator3      = 0;

    for (int c = colIdx - radius; c < colIdx + radius + 2; c++)
    {
        for (int r = rowIdx - radius; r < rowIdx + radius + 2; r++)
        {
            int t0 = std::abs(c - colIdx), t1 = cuda::abs(r - rowIdx);
            int t2 = std::abs(c - (colIdx + 1)), t3 = cuda::abs(r - (rowIdx + 1));
            int squared_dis0 = t0 * t0 + t1 * t1;
            int squared_dis1 = t2 * t2 + t1 * t1;
            int squared_dis2 = t0 * t0 + t3 * t3;
            int squared_dis3 = t2 * t2 + t3 * t3;

            if (!(squared_dis0 <= squared_radius || squared_dis1 <= squared_radius || squared_dis2 <= squared_radius
                  || squared_dis3 <= squared_radius))
            {
                continue;
            }

            work_type curr      = cuda::StaticCast<float>(src(batch_idx, r, c));
            work_type currColor = cuda::StaticCast<float>(srcColor(batch_idx, r, c));

            if (squared_dis0 <= squared_radius)
            {
                float e_space       = squared_dis0 * space_coefficient;
                float one_norm_size = norm1(currColor - centerColor0);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator0 += weight;
                numerator0 += weight * curr;
            }

            if (squared_dis1 <= squared_radius)
            {
                float e_space       = squared_dis1 * space_coefficient;
                float one_norm_size = norm1(currColor - centerColor1);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator1 += weight;
                numerator1 = numerator1 + (weight * curr);
            }

            if (squared_dis2 <= squared_radius)
            {
                float e_space       = squared_dis2 * space_coefficient;
                float one_norm_size = norm1(currColor - centerColor2);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator2 += weight;
                numerator2 = numerator2 + (weight * curr);
            }

            if (squared_dis3 <= squared_radius)
            {
                float e_space       = squared_dis3 * space_coefficient;
                float one_norm_size = norm1(currColor - centerColor3);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator3 += weight;
                numerator3 = numerator3 + (weight * curr);
            }
        }
    }
    if (colIdx < columns && rowIdx < rows)
    {
        *dst.ptr(coord0.z, coord0.y, coord0.x) = nvcv::cuda::SaturateCast<T>(numerator0 / denominator0);
    }
    if (colIdx + 1 < columns && rowIdx < rows)
    {
        *dst.ptr(coord1.z, coord1.y, coord1.x) = nvcv::cuda::SaturateCast<T>(numerator1 / denominator1);
    }
    if (colIdx < columns && rowIdx + 1 < rows)
    {
        *dst.ptr(coord2.z, coord2.y, coord2.x) = nvcv::cuda::SaturateCast<T>(numerator2 / denominator2);
    }
    if (colIdx + 1 < columns && rowIdx + 1 < rows)
    {
        *dst.ptr(coord3.z, coord3.y, coord3.x) = nvcv::cuda::SaturateCast<T>(numerator3 / denominator3);
    }
}

template<typename T, template<typename> class Brd>
void JointBilateralFilterVarShapeCaller(const IImageBatchVarShapeDataStridedCuda &inData,
                                        const IImageBatchVarShapeDataStridedCuda &inColorData,
                                        const IImageBatchVarShapeDataStridedCuda &outData, int batch,
                                        const cuda::Tensor1DWrap<int>   &inDiameter,
                                        const cuda::Tensor1DWrap<float> &inSigmaColor,
                                        const cuda::Tensor1DWrap<float> &inSigmaSpace, hipStream_t stream)
{
    Ptr2dVarShapeNHWC<T> src(inData);
    Ptr2dVarShapeNHWC<T> srcColor(inData);
    Ptr2dVarShapeNHWC<T> dst(outData);
    using work_type = cuda::ConvertBaseTypeTo<float, T>;
    Brd<work_type>                                     brd(0, 0, cuda::SetAll<work_type>(0.0f));
    BorderReader<Ptr2dVarShapeNHWC<T>, Brd<work_type>> brdSrc(src, brd);
    BorderReader<Ptr2dVarShapeNHWC<T>, Brd<work_type>> brdSrcColor(srcColor, brd);
    Size2D                                             outMaxSize = outData.maxSize();
    dim3                                               block(8, 8);
    dim3 grid(divUp(outMaxSize.w, block.x * 2), divUp(outMaxSize.h, block.y * 2), batch);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif

    JointBilateralFilterVarShapeKernel<<<grid, block, 0, stream>>>(brdSrc, brdSrcColor, dst, inDiameter, inSigmaColor,
                                                                   inSigmaSpace);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
}

ErrorCode JointBilateralFilterVarShape::infer(const IImageBatchVarShapeDataStridedCuda &inData,
                                              const IImageBatchVarShapeDataStridedCuda &inColorData,
                                              const IImageBatchVarShapeDataStridedCuda &outData,
                                              const ITensorDataStridedCuda             &diameterData,
                                              const ITensorDataStridedCuda             &sigmaColorData,
                                              const ITensorDataStridedCuda &sigmaSpaceData, NVCVBorderType borderMode,
                                              hipStream_t stream)
{
    cuda_op::DataFormat input_format      = GetLegacyDataFormat(inData);
    cuda_op::DataFormat inputColor_format = GetLegacyDataFormat(inColorData);
    cuda_op::DataFormat output_format     = GetLegacyDataFormat(outData);

    if (input_format != output_format)
    {
        LOG_ERROR("Input data format (" << input_format << ") and output data format (" << output_format
                                        << ") must be the same.");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inputColor_format != output_format)
    {
        LOG_ERROR("InputColor data format (" << inputColor_format << ") and output data format (" << output_format
                                             << ") must be the same.");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if ((input_format != kNHWC) && (input_format != kHWC))
    {
        LOG_ERROR("Invalid DataFormat both Input and Output must be kHWC or kNHWC");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if ((inputColor_format != kNHWC) && (inputColor_format != kHWC))
    {
        LOG_ERROR("Invalid DataFormat both InputColor and Output must be kHWC or kNHWC");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!inData.uniqueFormat())
    {
        LOG_ERROR("Images in the input varshape must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!inColorData.uniqueFormat())
    {
        LOG_ERROR("Images in the inputColor varshape must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inData.uniqueFormat() != outData.uniqueFormat())
    {
        LOG_ERROR("Input and Output formats must be same input format ="
                  << helpers::GetLegacyDataType(inData.uniqueFormat())
                  << " output format = " << helpers::GetLegacyDataType(outData.uniqueFormat()));
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inColorData.uniqueFormat() != outData.uniqueFormat())
    {
        LOG_ERROR("InputColor and Output formats must be same input format ="
                  << helpers::GetLegacyDataType(inColorData.uniqueFormat())
                  << " output format = " << helpers::GetLegacyDataType(outData.uniqueFormat()));
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!(borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REPLICATE || borderMode == NVCV_BORDER_REFLECT
          || borderMode == NVCV_BORDER_WRAP || borderMode == NVCV_BORDER_REFLECT101))
    {
        LOG_ERROR("[Error] Invalid borderMode " << borderMode);
        return ErrorCode::INVALID_PARAMETER;
    }

    DataType data_type = GetLegacyDataType(outData.uniqueFormat());
    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32S
          || data_type == kCV_32F))
    {
        LOG_ERROR("[Error] Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType diameter_data_type = GetLegacyDataType(diameterData.dtype());
    if (diameter_data_type != kCV_32S)
    {
        LOG_ERROR("[Error] Invalid diameterData DataType " << diameter_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType sigmaColor_data_type = GetLegacyDataType(sigmaColorData.dtype());
    if (sigmaColor_data_type != kCV_32F)
    {
        LOG_ERROR("[Error] Invalid sigmaColorData DataType " << sigmaColor_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType sigmaSpace_data_type = GetLegacyDataType(sigmaSpaceData.dtype());
    if (sigmaSpace_data_type != kCV_32F)
    {
        LOG_ERROR("[Error] Invalid sigmaSpaceData DataType " << sigmaSpace_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (inData.numImages() != outData.numImages())
    {
        LOG_ERROR("Input and Output data must have the same number of images (" << inData.numImages()
                                                                                << " != " << outData.numImages());
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int batch    = inData.numImages();
    int channels = inData.uniqueFormat().numChannels();
    if (channels > 4 || channels < 1)
    {
        LOG_ERROR("Invalid channel number ch = " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    // Create Tensor wrappers for parameter arrays
    cuda::Tensor1DWrap<int>   inDiameter(diameterData);
    cuda::Tensor1DWrap<float> inSigmaColor(sigmaColorData);
    cuda::Tensor1DWrap<float> inSigmaSpace(sigmaSpaceData);

    typedef void (*joint_bilateral_filter_var_shape_t)(
        const IImageBatchVarShapeDataStridedCuda &inData, const IImageBatchVarShapeDataStridedCuda &inColorData,
        const IImageBatchVarShapeDataStridedCuda &outData, int batch, const cuda::Tensor1DWrap<int> &inDiameter,
        const cuda::Tensor1DWrap<float> &inSigmaColor, const cuda::Tensor1DWrap<float> &inSigmaSpace,
        hipStream_t stream);

    // All templated functions instantiated here to remove one level of indirection that just hides the same lookup
    // table in 5 parts
    static const joint_bilateral_filter_var_shape_t funcs[5][6][4] = {
        {
         {JointBilateralFilterVarShapeCaller<uchar, BrdConstant>,
         JointBilateralFilterVarShapeCaller<uchar2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<uchar3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<uchar4, BrdConstant>},
         {JointBilateralFilterVarShapeCaller<char, BrdConstant>,
         JointBilateralFilterVarShapeCaller<char2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<char3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<char4, BrdConstant>},
         {JointBilateralFilterVarShapeCaller<ushort, BrdConstant>,
         JointBilateralFilterVarShapeCaller<ushort2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<ushort3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<ushort4, BrdConstant>},
         {JointBilateralFilterVarShapeCaller<short, BrdConstant>,
         JointBilateralFilterVarShapeCaller<short2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<short3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<short4, BrdConstant>},
         {JointBilateralFilterVarShapeCaller<int, BrdConstant>,
         JointBilateralFilterVarShapeCaller<int2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<int3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<int4, BrdConstant>},
         {JointBilateralFilterVarShapeCaller<float, BrdConstant>,
         JointBilateralFilterVarShapeCaller<float2, BrdConstant>,
         JointBilateralFilterVarShapeCaller<float3, BrdConstant>,
         JointBilateralFilterVarShapeCaller<float4, BrdConstant>},
         },
        {
         {JointBilateralFilterVarShapeCaller<uchar, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<uchar2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<uchar3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<uchar4, BrdReplicate>},
         {JointBilateralFilterVarShapeCaller<char, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<char2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<char3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<char4, BrdReplicate>},
         {JointBilateralFilterVarShapeCaller<ushort, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<ushort2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<ushort3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<ushort4, BrdReplicate>},
         {JointBilateralFilterVarShapeCaller<short, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<short2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<short3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<short4, BrdReplicate>},
         {JointBilateralFilterVarShapeCaller<int, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<int2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<int3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<int4, BrdReplicate>},
         {JointBilateralFilterVarShapeCaller<float, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<float2, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<float3, BrdReplicate>,
         JointBilateralFilterVarShapeCaller<float4, BrdReplicate>},
         },
        {
         {JointBilateralFilterVarShapeCaller<uchar, BrdReflect>,
         JointBilateralFilterVarShapeCaller<uchar2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<uchar3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<uchar4, BrdReflect>},
         {JointBilateralFilterVarShapeCaller<char, BrdReflect>,
         JointBilateralFilterVarShapeCaller<char2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<char3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<char4, BrdReflect>},
         {JointBilateralFilterVarShapeCaller<ushort, BrdReflect>,
         JointBilateralFilterVarShapeCaller<ushort2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<ushort3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<ushort4, BrdReflect>},
         {JointBilateralFilterVarShapeCaller<short, BrdReflect>,
         JointBilateralFilterVarShapeCaller<short2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<short3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<short4, BrdReflect>},
         {JointBilateralFilterVarShapeCaller<int, BrdReflect>, JointBilateralFilterVarShapeCaller<int2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<int3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<int4, BrdReflect>},
         {JointBilateralFilterVarShapeCaller<float, BrdReflect>,
         JointBilateralFilterVarShapeCaller<float2, BrdReflect>,
         JointBilateralFilterVarShapeCaller<float3, BrdReflect>,
         JointBilateralFilterVarShapeCaller<float4, BrdReflect>},
         },
        {
         {JointBilateralFilterVarShapeCaller<uchar, BrdWrap>, JointBilateralFilterVarShapeCaller<uchar2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<uchar3, BrdWrap>, JointBilateralFilterVarShapeCaller<uchar4, BrdWrap>},
         {JointBilateralFilterVarShapeCaller<char, BrdWrap>, JointBilateralFilterVarShapeCaller<char2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<char3, BrdWrap>, JointBilateralFilterVarShapeCaller<char4, BrdWrap>},
         {JointBilateralFilterVarShapeCaller<ushort, BrdWrap>, JointBilateralFilterVarShapeCaller<ushort2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<ushort3, BrdWrap>,
         JointBilateralFilterVarShapeCaller<ushort4, BrdWrap>},
         {JointBilateralFilterVarShapeCaller<short, BrdWrap>, JointBilateralFilterVarShapeCaller<short2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<short3, BrdWrap>, JointBilateralFilterVarShapeCaller<short4, BrdWrap>},
         {JointBilateralFilterVarShapeCaller<int, BrdWrap>, JointBilateralFilterVarShapeCaller<int2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<int3, BrdWrap>, JointBilateralFilterVarShapeCaller<int4, BrdWrap>},
         {JointBilateralFilterVarShapeCaller<float, BrdWrap>, JointBilateralFilterVarShapeCaller<float2, BrdWrap>,
         JointBilateralFilterVarShapeCaller<float3, BrdWrap>, JointBilateralFilterVarShapeCaller<float4, BrdWrap>},
         },
        {
         {JointBilateralFilterVarShapeCaller<uchar, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<uchar2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<uchar3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<uchar4, BrdReflect101>},
         {JointBilateralFilterVarShapeCaller<char, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<char2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<char3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<char4, BrdReflect101>},
         {JointBilateralFilterVarShapeCaller<ushort, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<ushort2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<ushort3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<ushort4, BrdReflect101>},
         {JointBilateralFilterVarShapeCaller<short, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<short2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<short3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<short4, BrdReflect101>},
         {JointBilateralFilterVarShapeCaller<int, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<int2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<int3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<int4, BrdReflect101>},
         {JointBilateralFilterVarShapeCaller<float, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<float2, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<float3, BrdReflect101>,
         JointBilateralFilterVarShapeCaller<float4, BrdReflect101>},
         },
    };

    funcs[borderMode][data_type][channels - 1](inData, inColorData, outData, batch, inDiameter, inSigmaColor,
                                               inSigmaSpace, stream);
    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
